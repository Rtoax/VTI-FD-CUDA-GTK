#include "hip/hip_runtime.h"
//a#########################################################
//a##         2D Acoustic VTI Medium Forward 
//a##    
//a##  Ps :GPU(CUDA)  
//a##	 Copyright (C) RongTao, All right reserve.
//a##/*a***************************
//a##Function for VTI medium modeling,
//a##
//a## Ps:  the function of modeling following:
//a##      
//a##          du/dt_=1/rho*dp/dx_ , 
//a##          dv/dt_=1/rho*dp/dy_ , 
//a##          dw/dt_=1/rho*dq/dz_ ,  
//a##          dp/dt_=rho*vpx^2*(du/dx_+dv/dy_)+rho*vp*vpn*dw/dz_ ,
//a##          dq/dt_=rho*vp*vpn*(du/dx_+dv/dy_)+rho*vp^2*dw/dz_ ,
//a##                     vpx^2=vp^2*(1+2*epsilu);
//a##                     vpn^2=vp^2*(1+2*deta);
//a##  
//a##*********a*******************/
//a##
//a##                                     Rong Tao 
//a##                            
//a#########################################################
#include<stdio.h>
#include<malloc.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include<hip/hip_runtime.h>

#define pi 3.141592653

__device__ float d0;

__device__ int mm=4;

__constant__ float c[4]={1.196289,-0.0797526,0.009570313,-0.0006975447};

void check_gpu_error2d (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
/*************func*******************/
void pad_vv2d(int nx,int nz,int nnx,int nnz,int npd,float *ee)
{
     int ix,iz,id;
 
    for(id=0;id<nnx*nnz;id++)
     {
       ix=id/nnz;
       iz=id%nnz;
       if(ix<npd){
           ee[id]=ee[npd*nnz+iz];  //left
       }else if(ix>=nnx-npd){
           ee[id]=ee[(nnx-npd-1)*nnz+iz];//right
       }
     }
    for(id=0;id<nnx*nnz;id++)
     {
       ix=id/nnz;
       iz=id%nnz;
       if(iz<npd){
           ee[id]=ee[ix*nnz+npd];//up
       }else if(iz>=nnz-npd){
           ee[id]=ee[ix*nnz+nnz-npd-1];//down
       }
       //if(ee[id]==0){printf("ee[%d][%d]==0.0\n",ix,iz);exit(0);}
     }
}
/*************func*******************/
bool read_file2d(const char FN1[],const char FN2[],const char FN3[],
               int nx,int nz,int nnx,int nnz,float *vv,float *epsilu,float *deta,int npd)
{
		 int i,j,id,vmax=0.0;
		
		 FILE *fp1,*fp2,*fp3;
		 if((fp1=fopen(FN1,"rb"))==NULL){printf("error open <%s>!\n",FN1);return false;}
		 if((fp2=fopen(FN2,"rb"))==NULL){printf("error open <%s>!\n",FN2);return false;}
		 if((fp3=fopen(FN3,"rb"))==NULL){printf("error open <%s>!\n",FN3);return false;}
		 for(i=npd;i<nx+npd;i++)
		 {
			 for(j=npd;j<nz+npd;j++)
			 {
                            id=i*nnz+j;
				 fread(&vv[id],4L,1,fp1);if(vmax<vv[id])vmax=vv[id];
				 fread(&epsilu[id],4L,1,fp2);
				 fread(&deta[id],4L,1,fp3);
			 }
		 }
		 fclose(fp1);printf("vmax=%d\n",vmax);
		 fclose(fp2);
		 fclose(fp3);

      return true;
}
/********************func**********************/
__global__ void get_d02d(float dx_,float dz_,int nnx,int nnz,int npd,float *vp)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;
       if(id<1)d0=10.0*vp[nnx*nnz/2]*log(100000.0)/(2.0*npd*((dx_+dz_)/2.0));
}
/*************func*******************/
__global__ void initial_coffe2d(float dt_,int nn,float *coff1,float *coff2,float *acoff1,float *acoff2,int npd)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nn+2*npd)
            {
		 if(id<npd)
		 {   
			 coff1[id]=1.0/(1.0+(dt_*d0*pow((npd-0.5-id)/npd,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt_*d0*pow((npd-0.5-id)/npd,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt_*d0*pow(((npd-id)*1.0)/npd,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt_*d0*pow(((npd-id)*1.0)/npd,2.0))/2.0);

		 }else if(id>=npd&&id<npd+nn){

			 coff1[id]=1.0;
			 coff2[id]=1.0;

			 acoff1[id]=1.0;
			 acoff2[id]=1.0;

		 }else{

			 coff1[id]=1.0/(1.0+(dt_*d0*pow((0.5+id-nn-npd)/npd,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt_*d0*pow((0.5+id-nn-npd)/npd,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt_*d0*pow(((id-nn-npd)*1.0)/npd,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt_*d0*pow(((id-nn-npd)*1.0)/npd,2.0))/2.0);
		 }	
            }       
}
/*************func*******************/
__global__ void shot_record2d(int nnx, int nnz, int nx, int nz, int npd, int it, int nt_, float *P, float *shot)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nx)
            {
               shot[it+nt_*id]=P[npd+nnz*(id+npd)];
            }       
}
/*************func**************/    
__global__ void mute_directwave2d(int nx,int nt,float dt,float favg,
                     float dx,float dz,int fs,int ds,int zs,int is,
                     float *vp,float *epsilu,float *shot,int tt)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

    int mu_t,mu_nt;
    float mu_x,mu_z,mu_t0;

    int ix=id/nt;
    int it=id%nt;

   if(id<nx*nt)
   {
        mu_x=dx*abs(ix-fs-(is-1)*ds);
        mu_z=dz*zs;
        mu_t0=sqrtf(pow(mu_x,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilu[1]));
        mu_t=(int)(2.0/(dt*favg));
        mu_nt=(int)(mu_t0/dt)+mu_t+tt;

           if((it>(int)(mu_t0/dt)-tt)&&it<mu_nt)
              shot[id]=0.0;
   }
}
//a################################################################################
__global__ void add_source2d(float pfac,float xsn,float zsn,int nx,int nz,int nnx,int nnz,float dt_,float t,
                        float favg_,int wtype,int npd,int is,int ds,float *P,float *Q)
/*< generate ricker wavelet with time deley >*/
{
       int ixs,izs;
       float x_,xx_,tdelay,ts,source=0.0,fs; 
  
       tdelay=1.0/favg_;
       ts=t-tdelay;
       fs=xsn+(is-1)*ds;

	if(wtype==1)//ricker wavelet
	{
          x_=favg_*ts;
          xx_=x_*x_;
          source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));
	}else if(wtype==2){//derivative of gaussian
          x_=(-4)*favg_*favg_*pi*pi/log(0.1);
          source=(-2)*pi*pi*ts*exp(-x_*ts*ts);
        }else if(wtype==3){//derivative of gaussian
          x_=(-1)*favg_*favg_*pi*pi/log(0.1);
          source=exp(-x_*ts*ts);
        }

       if(t<=2*tdelay)
       {         
	     ixs = (int)(fs+0.5)+npd-1;
            izs = (int)(zsn+0.5)+npd-1;
            P[ixs*nnz+izs]+=pfac*source;
            Q[ixs*nnz+izs]+=pfac*source;
       }
}
/*******************func*********************/
__global__ void update_vel2d(int nx,int nz,int nnx,int nnz,int npd,float dt_,float dx_,float dz_,
                           float *u0,float *w0,float *u1,float *w1,float *P,float *Q,
                           float *coffx1,float *coffx2,float *coffz1,float *coffz2)
{
	int id=threadIdx.x+blockDim.x*blockIdx.x;

	int ix,iz,im;
	float dtx,dtz,xx,zz;

        ix=id/nnz;
        iz=id%nnz;

		 dtx=dt_/dx_;
		 dtz=dt_/dz_;
               if(id>=mm&&id<nnx*nnz-mm)
                 {
                   if(ix>=mm&&ix<(nnx-mm)&&iz>=mm&&iz<(nnz-mm))
                    {
                     xx=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                      {
                        xx+=c[im]*(P[id+(im+1)*nnz]-P[id-im*nnz]);
                        zz+=c[im]*(Q[id+im+1]      -Q[id-im]);
                      }
                     u1[id]=coffx2[ix]*u0[id]-coffx1[ix]*dtx*xx;
                     w1[id]=coffz2[iz]*w0[id]-coffz1[iz]*dtz*zz;
                   }
                 }
}
/*******************func***********************/
__global__ void update_stress2d(int nx,int nz,int nnx,int nnz,float dt_,float dx_,float dz_,
                           float *u1,float *w1,float *P,float *Q,float *vp,int npd,
                           float *px1,float *px0,float *pz1,float *pz0,float *qx1,float *qx0,float *qz1,float *qz0,
                           float *acoffx1,float *acoffx2,float *acoffz1,float *acoffz2,
                           float *deta,float *epsilu,int fs,int ds,int zs,int is,int SV)
{
    int id=threadIdx.x+blockDim.x*blockIdx.x;

	int im,ix,iz,rx,rz,R=15,r=4;
	float dtx,dtz, xx,zz,ee,dd;

        ix=id/nnz;
        iz=id%nnz;

               dtx=dt_/dx_;
		 dtz=dt_/dz_;
               if(id>=mm&&id<nnx*nnz-mm)
                 {
/************************i****************************************/
/************************iso circle start*************************/
                   rx=ix-(fs+(is-1)*ds+npd);
                   rz=iz-(zs+npd);
                   if(SV){
                       if((rx*rx+rz*rz)<=R*R){
                           if((rx*rx+rz*rz)<=r*r){
                               ee = 0.0;
                               dd = 0.0;
                           }else{
                               ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*epsilu[id];
                               dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+rz*rz)-r)*4.0/(R*3.0-1))))*deta[id]; 
                              }
                       }else{
                          ee=epsilu[id];
                          dd=deta[id];
                          }
                   }else{
                      ee=epsilu[id];
                      dd=deta[id];
                     }
/************************ iso circle end *************************/
/************************i****************************************/
                   if(ix>=mm&&ix<(nnx-mm)&&iz>=mm&&iz<(nnz-mm))
                     {
                     xx=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                       {
                        xx+=c[im]*(u1[id+im*nnz]-u1[id-(im+1)*nnz]);
                        zz+=c[im]*(w1[id+im]    -w1[id-im-1]);
                       }
                     px1[id]=acoffx2[ix]*px0[id]-acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
                     pz1[id]=acoffz2[iz]*pz0[id]-acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;
                     qx1[id]=acoffx2[ix]*qx0[id]-acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
                     qz1[id]=acoffz2[iz]*qz0[id]-acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

                     P[id]=px1[id]+pz1[id];
                     Q[id]=qx1[id]+qz1[id];
                   }
                 }
} 
//###################################model#######################################
extern "C"  void GPU_vti2dfd(int nx, int nz,int dx,int dz,int npd,int SV,
           const char FNv[],const char FNe[],const char FNd[],
           int favg,int ns,int fs,int ds,int zs,
           const char FNshot[],const char FNsnap[],int nt, int dt,int run_count)
{
  //int ite=0;
  //loop1:if(ite>0){printf("Please ensure parameter input accurately! \n");ite++;}

  float dx_,dz_,favg_,dt_,pfac;

  dx_=(float)dx;
  dz_=(float)dz;
  favg_=(float)favg;

  printf("\n##### model start #####\n");
  printf("#  nx=%2d, dx=%.2f, npd=%d\n",nx,dx_,npd);
  printf("#  nz=%2d, dz=%.2f, SV=%d\n",nz,dz_,SV);
  printf("#     vel=<%s>\n",FNv);
  printf("#  epsilu=<%s>\n",FNe);
  printf("#    deta=<%s>\n",FNd);
  printf("#  favg=%.2f\n",favg_);
  printf("#  ns=%3d\n",ns);
  printf("#  fs=%3d\n",fs);
  printf("#  ds=%3d\n",ds);
  printf("#  zs=%3d\n",zs);
  printf("#    shot=<%s>\n",FNshot);
  printf("#    snap=<%s>\n",FNsnap);


	 FILE *fpsnap, *fpshot;
        fpshot=fopen(FNshot,"wb");
        fpsnap=fopen(FNsnap,"wb");


	int is, it, nnx, nnz, nt_, wtype;
	float *v, *e, *d, t;
	float *vp, *epsilu, *deta;
	float *u0, *u1, *px0, *qx0, *px1, *qx1;
       float *w0, *w1, *pz0, *qz0, *pz1, *qz1;
	float *P, *Q, *shot_Dev, *shot_Hos;

       float *coffx1,*coffx2,*coffz1,*coffz2,*acoffx1,*acoffx2,*acoffz1,*acoffz2;

       clock_t start, end;

          wtype=1;
	   nt_=nt;    
          dt_=(float)(dt*1.0/1000000);
          pfac=10.0;

          nnx=nx+2*npd;
          nnz=nz+2*npd;

    	 v=(float*)malloc(nnz*nnx*sizeof(float));
    	 e=(float*)malloc(nnz*nnx*sizeof(float));
    	 d=(float*)malloc(nnz*nnx*sizeof(float));
    	 shot_Hos=(float*)malloc(nt_*nx*sizeof(float));
        if(read_file2d(FNv,FNe,FNd,nx,nz,nnx,nnz,v,e,d,npd))printf("Read file done !\n");
        else{printf("Read file error !\n");return;}
        pad_vv2d(nx,nz,nnx,nnz,npd,e);
        pad_vv2d(nx,nz,nnx,nnz,npd,d);
        pad_vv2d(nx,nz,nnx,nnz,npd,v); 

        hipSetDevice(0);// initialize device, default device=0;
	 if(run_count==0)check_gpu_error2d("Failed to initialize device!");
 
/****************************/
        hipMalloc(&vp, nnz*nnx*sizeof(float));
        hipMalloc(&epsilu, nnz*nnx*sizeof(float));
        hipMalloc(&deta, nnz*nnx*sizeof(float));
	 hipMemcpy(vp, v, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(epsilu, e, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(deta, d, nnz*nnx*sizeof(float), hipMemcpyHostToDevice);
/****************************/
        hipMalloc(&u0, nnz*nnx*sizeof(float));    hipMalloc(&u1, nnz*nnx*sizeof(float));
        hipMalloc(&w0, nnz*nnx*sizeof(float));    hipMalloc(&w1, nnz*nnx*sizeof(float));

        hipMalloc(&P, nnz*nnx*sizeof(float));     hipMalloc(&Q, nnz*nnx*sizeof(float));

        hipMalloc(&px0, nnz*nnx*sizeof(float));   hipMalloc(&px1, nnz*nnx*sizeof(float));
        hipMalloc(&pz0, nnz*nnx*sizeof(float));   hipMalloc(&pz1, nnz*nnx*sizeof(float));
        hipMalloc(&qx0, nnz*nnx*sizeof(float));   hipMalloc(&qx1, nnz*nnx*sizeof(float));
        hipMalloc(&qz0, nnz*nnx*sizeof(float));   hipMalloc(&qz1, nnz*nnx*sizeof(float));

        hipMalloc(&coffx1, nnx*sizeof(float));     hipMalloc(&coffx2, nnx*sizeof(float));
        hipMalloc(&coffz1, nnz*sizeof(float));     hipMalloc(&coffz2, nnz*sizeof(float));
        hipMalloc(&acoffx1, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
        hipMalloc(&acoffz1, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

        hipMalloc(&shot_Dev, nx*nt_*sizeof(float));

	 if(run_count==0)check_gpu_error2d("Failed to allocate memory for variables!");


        get_d02d<<<1, 1>>>(dx_, dz_, nnx, nnz, npd, vp);

        initial_coffe2d<<<(nnx+511)/512, 512>>>(dt_,nx,coffx1,coffx2,acoffx1,acoffx2,npd);
        initial_coffe2d<<<(nnz+511)/512, 512>>>(dt_,nz,coffz1,coffz2,acoffz1,acoffz2,npd);

        printf("--------------------------------------------------------\n");
        printf("---   \n");   
        start = clock(); 

   for(is=1;is<=ns;is++)	
    {     
         printf("---   IS=%3d  \n",is);
     hipMemset(u0, 0, nnz*nnx*sizeof(float));     hipMemset(u1, 0, nnz*nnx*sizeof(float));
     hipMemset(w0, 0, nnz*nnx*sizeof(float));     hipMemset(w1, 0, nnz*nnx*sizeof(float));

     hipMemset(P, 0, nnz*nnx*sizeof(float));      hipMemset(Q, 0, nnz*nnx*sizeof(float));

     hipMemset(px0, 0, nnz*nnx*sizeof(float));    hipMemset(px1, 0, nnz*nnx*sizeof(float));
     hipMemset(pz0, 0, nnz*nnx*sizeof(float));    hipMemset(pz1, 0, nnz*nnx*sizeof(float));
     hipMemset(qx0, 0, nnz*nnx*sizeof(float));    hipMemset(qx1, 0, nnz*nnx*sizeof(float));
     hipMemset(qz0, 0, nnz*nnx*sizeof(float));    hipMemset(qz1, 0, nnz*nnx*sizeof(float));

     hipMemset(shot_Dev, 0, nt_*nx*sizeof(float));

     for(it=0,t=dt_;it<nt_;it++,t+=dt_)
     { 
     // if(it%100==0&&is==1)printf("---   is===%d   it===%d\n",is,it);

	 add_source2d<<<1,1>>>(pfac,fs,zs,nx,nz,nnx,nnz,dt_,t,favg_,wtype,npd,is,ds,P,Q);     
        update_vel2d<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,npd,dt_,dx_,dz_,u0,w0,u1,w1,P,Q,coffx1,coffx2,coffz1,coffz2);
        update_stress2d<<<(nnx*nnz+511)/512, 512>>>(nx,nz,nnx,nnz,dt_,dx_,dz_,u1,w1,P,Q,vp,npd,px1,px0,pz1,pz0,qx1,qx0,qz1,qz0,
                                                  acoffx1,acoffx2,acoffz1,acoffz2,deta,epsilu,fs,ds,zs,is,SV);
        u0=u1; w0=w1; px0=px1; pz0=pz1; qx0=qx1; qz0=qz1; 

        shot_record2d<<<(nx+511)/512, 512>>>(nnx, nnz, nx, nz, npd, it, nt_, P, shot_Dev);


           if((is==1)&&(it%50==0))
            {
	       hipMemcpy(e, P, nnz*nnx*sizeof(float), hipMemcpyDeviceToHost);
              fseek(fpsnap,(int)(it/50)*(nnx)*(nnz)*4L,0);
              fwrite(e,4L,nnx*nnz,fpsnap);
            }
     }//it loop end
      mute_directwave2d<<<(nx*nt_+511)/512, 512>>>(nx,nt_,dt_,favg_,dx_,dz_,fs,ds,zs,is,vp,epsilu,shot_Dev,100);
      hipMemcpy(shot_Hos, shot_Dev, nt_*nx*sizeof(float), hipMemcpyDeviceToHost);
      fseek(fpshot,(is-1)*nt_*nx*sizeof(float),0);
      fwrite(shot_Hos,sizeof(float),nt_*nx,fpshot);

    }

    end = clock();
/*********IS Loop end*********/ 		     
   printf("---   The forward is over    \n"); 
   printf("---   Complete!!!!!!!!! \n");  
   printf("total %d shots: %f (s)\n", ns, ((float)(end-start))/CLOCKS_PER_SEC);

/***********close************/ 
          fclose(fpsnap);   fclose(fpshot);
/***********free*************/ 
       hipFree(coffx1);       hipFree(coffx2);
       hipFree(coffz1);       hipFree(coffz2);
       hipFree(acoffx1);      hipFree(acoffx2);
       hipFree(acoffz1);      hipFree(acoffz2);

       hipFree(u0);           hipFree(u1);
       hipFree(w0);           hipFree(w1);

       hipFree(P);            hipFree(Q);

       hipFree(px0);          hipFree(px1);
       hipFree(pz0);          hipFree(pz1);
       hipFree(qx0);          hipFree(qx1);
       hipFree(qz0);          hipFree(qz1);

       hipFree(shot_Dev);

       hipFree(vp);
       hipFree(epsilu);
       hipFree(deta);

       
/***************host free*****************/
	free(v);	free(e);	free(d);
       free(shot_Hos);


     //  exit(0);

}
