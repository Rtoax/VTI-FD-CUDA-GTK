#include "hip/hip_runtime.h"
//a#########################################################
//a##         3D Acoustic VTI Medium Forward 
//a##    
//a##  Ps :GPU(CUDA)  
//a##
//a##/*a***************************
//a##Function for VTI medium modeling,
//a##
//a## Ps:  the function of modeling following:
//a##      
//a##          du/dt_=1/rho*dp/dx_ , 
//a##          dv/dt_=1/rho*dp/dy_ , 
//a##          dw/dt_=1/rho*dq/dz_ ,  
//a##          dp/dt_=rho*vpx^2*(du/dx_+dv/dy_)+rho*vp*vpn*dw/dz_ ,
//a##          dq/dt_=rho*vp*vpn*(du/dx_+dv/dy_)+rho*vp^2*dw/dz_ ,
//a##                     vpx^2=vp^2*(1+2*epsilu);
//a##                     vpn^2=vp^2*(1+2*deta);
//a##  
//a##*********a*******************/
//a##
//a##                                     Rong Tao 
//a##                            
//a#########################################################
#include<stdio.h>
#include<malloc.h>
#include<math.h>
#include<stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define pi 3.141592653

#define BlockSize1 16// tile size in 1st-axis
#define BlockSize2 16// tile size in 2nd-axis

#define mm 4

__device__ float d0;

__constant__ float c[mm]={1.196289,-0.0797526,0.009570313,-0.0006975447};

//a################################################################################
void check_gpu_error3d (const char *msg) 
/*< check GPU errors >*/
{
    hipError_t err = hipGetLastError ();
    if (hipSuccess != err) { 
	printf("Cuda error: %s: %s\n", msg, hipGetErrorString (err)); 
	exit(0);   
    }
}
//a################################################################################
__global__ void add_source3d(float pfac,int fsx,int fsy,int fsz,int nx,int ny,int nz,int nnx,int nny,int nnz,float dt_,float t,
                        float favg_,int wtype,int npml,int is,int dsx,int dsy,int dsz,float *P,float *Q)
/*< generate ricker wavelet with time deley >*/
{
       int ixs,iys,izs;
       float x_,xx_,tdelay,ts,source=0.0,sx,sy,sz; 
  
       tdelay=1.0/favg_;
       ts=t-tdelay;

       sx=fsx+is*dsx;
       sy=fsy+is*dsy;
       sz=fsz+is*dsz;

	if(wtype==1)//ricker wavelet
	{
          x_=favg_*ts;
          xx_=x_*x_;
          source=(1-2*pi*pi*(xx_))*exp(-(pi*pi*xx_));
	}else if(wtype==2){//derivative of gaussian
          x_=(-4)*favg_*favg_*pi*pi/log(0.1);
          source=(-2)*pi*pi*ts*exp(-x_*ts*ts);
        }else if(wtype==3){//derivative of gaussian
          x_=(-1)*favg_*favg_*pi*pi/log(0.1);
          source=exp(-x_*ts*ts);
        }

       if(t<=2*tdelay)
       {         
	     ixs = sx+npml-1;
	     iys = sy+npml-1;
            izs = sz+npml-1;
            P[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
            Q[izs+ixs*nnz+iys*nnz*nnx]+=pfac*source;
       }
}
/*******************func*********************/
__global__ void update_vel3d(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float dt_,float dx_,float dy_,float dz_,
                           float *u0,float *v0,float *w0,float *u1,float *v1,float *w1,float *P,float *Q,
                           float *coffx1,float *coffx2,float *coffy1,float *coffy2,float *coffz1,float *coffz2)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im;
	float dtx,dty,dtz,xx,yy,zz;

		 dtx=dt_/dx_;
		 dty=dt_/dy_;
		 dtz=dt_/dz_;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nny*nnz-mm)
                 {
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                    {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                      {
                        yy+=c[im]*(P[id+(im+1)*nnz*nnx] - P[id-im*nnz*nnx]);
                        xx+=c[im]*(P[id+(im+1)*nnz]     - P[id-im*nnz]);
                        zz+=c[im]*(Q[id+im+1]           - Q[id-im]);
                      }
                     u1[id]=coffx2[ix]*u0[id]-coffx1[ix]*dtx*xx;
                     v1[id]=coffy2[iy]*v0[id]-coffy1[iy]*dty*yy;
                     w1[id]=coffz2[iz]*w0[id]-coffz1[iz]*dtz*zz;
                   }
                 }
        }  



}
/*******************func***********************/
__global__ void update_stress3d(int nx,int ny,int nz,int nnx,int nny,int nnz,float dt_,float dx_,float dy_,float dz_,
                           float *u1,float *v1,float *w1,float *P,float *Q,float *vp,int npml,
                           float *px1,float *px0,float *py1,float *py0,float *pz1,float *pz0,
                           float *qx1,float *qx0,float *qy1,float *qy0,float *qz1,float *qz0,
                           float *acoffx1,float *acoffx2,float *acoffy1,float *acoffy2,float *acoffz1,float *acoffz2,
                           float *deta,float *epsilu,int fsx,int dsx,int fsy,int dsy,int fsz,int dsz,int is,int SV)
{
    const int iz = blockIdx.x * blockDim.x + threadIdx.x;//0--nz's thread:iz
    const int ix = blockIdx.y * blockDim.y + threadIdx.y;//0--nx's thread:ix

       int id,iy,im,rx,ry,rz,R=15,r=4;
	float dtx,dty,dtz,xx,yy,zz,ee,dd;

		 dtx=dt_/dx_;
		 dty=dt_/dy_;
		 dtz=dt_/dz_;

       for(iy=0;iy<nny;iy++)
        {
               id=iz+ix*nnz+iy*nnz*nnx;
               if(id>=mm&&id<nnx*nnz*nny-mm)
                 {
/************************i****************************************/
/************************iso circle start*************************/
                   rx=ix-(fsx+is*dsx+npml-1);
                   ry=iy-(fsy+is*dsy+npml-1);
                   rz=iz-(fsz+is*dsz+npml-1);
                   if(SV){
                       if((rx*rx+ry*ry+rz*rz)<=R*R){
                           if((rx*rx+ry*ry+rz*rz)<=r*r){
                               ee = 0.0;
                               dd = 0.0;
                           }else{
                               ee = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*epsilu[id];
                               dd = 0.5*(1-cos(pi*((sqrtf(rx*rx+ry*ry+rz*rz)-r)*4.0/(R*3.0-1))))*deta[id]; 
                              }
                       }else{
                          ee=epsilu[id];
                          dd=deta[id];
                          }
                   }else{
                      ee=epsilu[id];
                      dd=deta[id];
                     }
/************************ iso circle end *************************/
/************************i****************************************/
                   if(ix>=mm&&ix<(nnx-mm)&&iy>=mm&&iy<(nny-mm)&&iz>=mm&&iz<(nnz-mm))
                     {
                     xx=0.0;
                     yy=0.0;
                     zz=0.0;
	             for(im=0;im<mm;im++)
                       {
                        yy+=c[im]*(v1[id+im*nnz*nnx] - v1[id-(im+1)*nnz*nnx]);
                        xx+=c[im]*(u1[id+im*nnz]     - u1[id-(im+1)*nnz]);
                        zz+=c[im]*(w1[id+im]         - w1[id-im-1]);
                       }
                     px1[id]=acoffx2[ix]*px0[id] - acoffx1[ix]*vp[id]*vp[id]*(1+2*ee)*dtx*xx;
                     py1[id]=acoffy2[iy]*py0[id] - acoffy1[iy]*vp[id]*vp[id]*(1+2*ee)*dty*yy;
                     pz1[id]=acoffz2[iz]*pz0[id] - acoffz1[iz]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtz*zz;

                     qx1[id]=acoffx2[ix]*qx0[id] - acoffx1[ix]*vp[id]*vp[id]*sqrtf(1+2*dd)*dtx*xx;
                     qy1[id]=acoffy2[iy]*qy0[id] - acoffy1[iy]*vp[id]*vp[id]*sqrtf(1+2*dd)*dty*yy;
                     qz1[id]=acoffz2[iz]*qz0[id] - acoffz1[iz]*vp[id]*vp[id]*dtz*zz;

                     P[id]=px1[id]+py1[id]+pz1[id];
                     Q[id]=qx1[id]+qy1[id]+qz1[id];
                   }
                 }
         }
}                      
/********************func**********************/
__global__ void get_d03d(float dx_,float dy_,float dz_,int nnx,int nny,int nnz,int npml,float *vp)
{
   d0=10.0*vp[nny*nnx*nnz/2]*log(100000.0)/(2.0*npml*((dx_+dy_+dz_)/3.0));
}
/*************func*******************/
void pad_vv3d(int nx,int ny,int nz,int nnx,int nny,int nnz,int npml,float *ee)
{
     int ix,iy,iz,id;
 
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(ix<npml){
                                ee[id]=ee[iz+npml*nnz+iy*nnz*nnx];  //left
                             }else if(ix>=nnx-npml){
                                ee[id]=ee[iz+(nnx-npml-1)*nnz+iy*nnz*nnx];//right
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iy<npml){
                                ee[id]=ee[iz+ix*nnz+npml*nnz*nnx];  //front
                             }else if(iy>=nny-npml){
                                ee[id]=ee[iz+ix*nnz+(nny-npml-1)*nnz*nnx];//back
                                 }
			 }
		 }
	    for(iy=0;iy<nny;iy++)
		 for(ix=0;ix<nnx;ix++)
		 {
			 for(iz=0;iz<nnz;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;

                             if(iz<npml){
                                ee[id]=ee[npml+ix*nnz+iy*nnz*nnx];  //up
                             }else if(iz>=nnz-npml){
                                ee[id]=ee[nnz-npml-1+ix*nnz+iy*nnz*nnx];//down
                                 }
			 }
		 }

}
/*************func*******************/
void read_file3d(const char FNv[],const char FNe[],const char FNd[],int nx,int ny,int nz,
               int nnx,int nny,int nnz,float *vv,float *epsilu,float *deta,int npml)
{
		 int ix,iy,iz,id;
		
		 FILE *fp1,*fp2,*fp3;
		 if((fp1=fopen(FNv,"rb"))==NULL)printf("error open <%s>!\n",FNv);
		 if((fp2=fopen(FNe,"rb"))==NULL)printf("error open <%s>!\n",FNe);
		 if((fp3=fopen(FNd,"rb"))==NULL)printf("error open <%s>!\n",FNd);

	    for(iy=npml;iy<ny+npml;iy++)
		 for(ix=npml;ix<nx+npml;ix++)
		 {
			 for(iz=npml;iz<nz+npml;iz++)
			 {
                             id=iz+ix*nnz+iy*nnz*nnx;
				 fread(&vv[id],4L,1,fp1);//vv[id]=3000.0;
				 fread(&epsilu[id],4L,1,fp2);
				 fread(&deta[id],4L,1,fp3);
			 }
		 }
		 fclose(fp1);
		 fclose(fp2);
		 fclose(fp3);
}
/*************func*******************/
__global__ void initial_coffe3d(float dt_,int nn,float *coff1,float *coff2,float *acoff1,float *acoff2,int npml)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

           if(id<nn+2*npml)
            {
		 if(id<npml)
		 {   
			 coff1[id]=1.0/(1.0+(dt_*d0*pow((npml-0.5-id)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt_*d0*pow((npml-0.5-id)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt_*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt_*d0*pow(((npml-id)*1.0)/npml,2.0))/2.0);

		 }else if(id>=npml&&id<npml+nn){

			 coff1[id]=1.0;
			 coff2[id]=1.0;

			 acoff1[id]=1.0;
			 acoff2[id]=1.0;

		 }else{

			 coff1[id]=1.0/(1.0+(dt_*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);
			 coff2[id]=coff1[id]*(1.0-(dt_*d0*pow((0.5+id-nn-npml)/npml,2.0))/2.0);

			 acoff1[id]=1.0/(1.0+(dt_*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
			 acoff2[id]=acoff1[id]*(1.0-(dt_*d0*pow(((id-nn-npml)*1.0)/npml,2.0))/2.0);
		 }	
            }       
}
/*************func*******************/
__global__ void shot_record3d(int nnx,int nny, int nnz, int nx,int ny, int nz, int npml, int it, int nt, float *P, float *shot)
{		
	 int id=threadIdx.x+blockDim.x*blockIdx.x;

        int ix=id%nx;
        int iy=id/nx;

           if(id<nx*ny)
            {
               shot[it+nt*ix+nt*nx*iy]=P[npml+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
            }       
}

/*************func**************/ 
void window3d(float *a, float *b, int nz, int nx, int ny, int nnz, int nnx, int npml)
/*< window a 3d subvolume >*/
{
	int iz, ix, iy;
	
	for(iy=0; iy<ny; iy++)
	for(ix=0; ix<nx; ix++)
	for(iz=0; iz<nz; iz++)
	{
		a[iz+nz*ix+nz*nx*iy]=b[(iz+npml)+nnz*(ix+npml)+nnz*nnx*(iy+npml)];
	}
}
/*************func**************/    
__global__ void mute_directwave3d(int nx,int ny,int nt,float dt_,float favg_, float dx_,float dy_,float dz_,int fsx,int fsy,int dsx,int dsy,
                                int fsz,int is, float *vp,float *epsilu,float *shot,int tt)
{

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int id,it;
    int mu_t,mu_nt;
    float mu_x,mu_y,mu_z,mu_t0;

       for(it=0;it<nt;it++)
        {
          id=it+ix*nt+iy*nx*nt;
          if(ix<nx&&iy<ny&&it<nt)
            {
              mu_x=dx_*abs(ix-fsx-(is)*dsx);
              mu_y=dy_*abs(iy-fsy-(is)*dsy);
              mu_z=dz_*fsz;
              mu_t0=sqrtf(pow(mu_x,2)+pow(mu_y,2)+pow(mu_z,2))/(vp[1]*sqrtf(1+2*epsilu[1]));
              mu_t=(int)(2.0/(dt_*favg_));
              mu_nt=(int)(mu_t0/dt_)+mu_t+tt;

                 if(it<mu_nt)
                    shot[id]=0.0;
            }
        }
}
//a########################################################################
extern "C"  void GPU_vti3dfd(int nx, int ny, int nz,int dx,int dy,int dz,int npml,int SV,
           const char FNv[],const char FNe[],const char FNd[],
           int favg,int ns,int fsx,int dsx,int fsy,int dsy,int fsz,int dsz,
           const char FNshot[],const char FNsnap[],int nt, int dt,int run_count)
{
	int is, it, nnx, nny, nnz,  wtype;
	float dx_, dy_, dz_, dt_, t, pfac, favg_;

	float *v, *e, *d;
	float *vp, *epsilu, *deta;
	float *s_u0, *s_u1, *s_px0, *s_qx0, *s_px1, *s_qx1;
	float *s_v0, *s_v1, *s_py0, *s_qy0, *s_py1, *s_qy1;
       float *s_w0, *s_w1, *s_pz0, *s_qz0, *s_pz1, *s_qz1;
	float *s_P, *s_Q, *shot_Dev, *shot_Hos;

       float *coffx1,*coffx2,*coffy1,*coffy2,*coffz1,*coffz2;
       float *acoffx1,*acoffx2,*acoffy1,*acoffy2,*acoffz1,*acoffz2;

       clock_t start, end;
/*************wavelet\boundary**************/
          wtype=1;
/********** dat document ***********/

/********aaa************/  
	 FILE *fpsnap, *fpshot;
        fpshot=fopen(FNshot,"wb");
        fpsnap=fopen(FNsnap,"wb");

 
/********* parameters *************/

     pfac=10.0;
  
       dx_=(float)dx;
       dy_=(float)dy;
       dz_=(float)dz;
       favg_=(float)favg;
       dt_=(float)(dt*1.0/1000000);
        
/*************v***************/ 
          nnx=nx+2*npml;
          nny=ny+2*npml;
          nnz=nz+2*npml;
/************a*************/
  printf("\n##### model start #####\n");
  printf("#  nx=%2d, dx=%.2f, npd=%d\n",nx,dx_,npml);
  printf("#  ny=%2d, dy=%.2f, \n",ny,dy_);
  printf("#  nz=%2d, dz=%.2f, SV=%d\n",nz,dz_,SV);
  printf("#     vel=<%s>\n",FNv);
  printf("#  epsilu=<%s>\n",FNe);
  printf("#    deta=<%s>\n",FNd);
  printf("#  favg=%.2f\n",favg_);
  printf("#  ns=%3d\n",ns);
  printf("#  fsx=%3d, fsy=%3d, fsz=%3d, \n",fsx,fsy,fsz);
  printf("#  dsx=%3d, dsy=%3d, dsz=%3d, \n",dsx,dsy,dsz);
  printf("#    shot=<%s>\n",FNshot);
  printf("#    snap=<%s>\n",FNsnap);

    	 v=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 e=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 d=(float*)malloc(nnz*nnx*nny*sizeof(float));
    	 shot_Hos=(float*)malloc(nt*nx*ny*sizeof(float));
        read_file3d(FNv,FNe,FNd,nx,ny,nz,nnx,nny,nnz,v,e,d,npml);
/****************************/
        pad_vv3d(nx,ny,nz,nnx,nny,nnz,npml,e);
        pad_vv3d(nx,ny,nz,nnx,nny,nnz,npml,d);
        pad_vv3d(nx,ny,nz,nnx,nny,nnz,npml,v);

        hipSetDevice(0);// initialize device, default device=0;
	 if(run_count==0)check_gpu_error3d("Failed to initialize device!");

	dim3 Xdimg, dimg, dimb;
	Xdimg.x=(nnx+BlockSize1-1)/BlockSize1;
	Xdimg.y=(nny+BlockSize2-1)/BlockSize2;
	dimg.x=(nnz+BlockSize1-1)/BlockSize1;
	dimg.y=(nnx+BlockSize2-1)/BlockSize2;
	dimb.x=BlockSize1;
	dimb.y=BlockSize2;
/****************************/
        hipMalloc(&vp, nnz*nnx*nny*sizeof(float));
        hipMalloc(&epsilu, nnz*nnx*nny*sizeof(float));
        hipMalloc(&deta, nnz*nnx*nny*sizeof(float));
	 hipMemcpy(vp, v, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(epsilu, e, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
	 hipMemcpy(deta, d, nnz*nnx*nny*sizeof(float), hipMemcpyHostToDevice);
/****************************/
        hipMalloc(&s_u0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_u1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_v0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_v1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_w0, nnz*nnx*nny*sizeof(float));    hipMalloc(&s_w1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_P, nnz*nnx*nny*sizeof(float));     hipMalloc(&s_Q, nnz*nnx*nny*sizeof(float));

        hipMalloc(&s_px0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_px1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_py0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_py1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_pz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_pz1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qx0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qx1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qy0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qy1, nnz*nnx*nny*sizeof(float));
        hipMalloc(&s_qz0, nnz*nnx*nny*sizeof(float));   hipMalloc(&s_qz1, nnz*nnx*nny*sizeof(float));

        hipMalloc(&coffx1, nnx*sizeof(float));     hipMalloc(&coffx2, nnx*sizeof(float));
        hipMalloc(&coffy1, nnx*sizeof(float));     hipMalloc(&coffy2, nnx*sizeof(float));
        hipMalloc(&coffz1, nnz*sizeof(float));     hipMalloc(&coffz2, nnz*sizeof(float));
        hipMalloc(&acoffx1, nnx*sizeof(float));    hipMalloc(&acoffx2, nnx*sizeof(float));
        hipMalloc(&acoffy1, nnx*sizeof(float));    hipMalloc(&acoffy2, nnx*sizeof(float));
        hipMalloc(&acoffz1, nnz*sizeof(float));    hipMalloc(&acoffz2, nnz*sizeof(float));

        hipMalloc(&shot_Dev, nx*ny*nt*sizeof(float));
/******************************/
	 if(run_count==0)check_gpu_error3d("Failed to allocate memory for variables!");

        get_d03d<<<1, 1>>>(dx_,dy_,dz_,nnx,nny,nnz,npml,vp);
        initial_coffe3d<<<(nnx+511)/512, 512>>>(dt_,nx,coffx1,coffx2,acoffx1,acoffx2,npml);
        initial_coffe3d<<<(nny+511)/512, 512>>>(dt_,ny,coffy1,coffy2,acoffy1,acoffy2,npml);
        initial_coffe3d<<<(nnz+511)/512, 512>>>(dt_,nz,coffz1,coffz2,acoffz1,acoffz2,npml);



        printf("--------------------------------------------------------\n");
        printf("---   \n");   
        start = clock();                                  
/**********IS Loop start*******/
   for(is=0;is<ns;is++)	
    {     
       //  printf("---   IS=%3d  \n",is);

     hipMemset(s_u0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_u1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_v0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_v1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_w0, 0, nnz*nnx*nny*sizeof(float));     hipMemset(s_w1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_P, 0, nnz*nnx*nny*sizeof(float));      hipMemset(s_Q, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(s_px0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_px1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_py0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_py1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_pz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_pz1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qx0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qx1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qy0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qy1, 0, nnz*nnx*nny*sizeof(float));
     hipMemset(s_qz0, 0, nnz*nnx*nny*sizeof(float));    hipMemset(s_qz1, 0, nnz*nnx*nny*sizeof(float));

     hipMemset(shot_Dev, 0, nt*nx*ny*sizeof(float));

     for(it=0,t=dt_;it<nt;it++,t+=dt_)
     { 
      if(it%100==0)printf("---   IS===%d   it===%d\n",is,it);
        add_source3d<<<1,1>>>(pfac,fsx,fsy,fsz,nx,ny,nz,nnx,nny,nnz,dt_,t,favg_,wtype,npml,is,dsx,dsy,dsz,s_P,s_Q);
        update_vel3d<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,npml,dt_,dx_,dy_,dz_,
                                 s_u0,s_v0,s_w0,s_u1,s_v1,s_w1,s_P,s_Q,coffx1,coffx2,coffy1,coffy2,coffz1,coffz2);
        update_stress3d<<<dimg,dimb>>>(nx,ny,nz,nnx,nny,nnz,dt_,dx_,dy_,dz_,s_u1,s_v1,s_w1,s_P,s_Q,vp,npml,
                                     s_px1,s_px0,s_py1,s_py0,s_pz1,s_pz0,s_qx1,s_qx0,s_qy1,s_qy0,s_qz1,s_qz0,
                                     acoffx1,acoffx2,acoffy1,acoffy2,acoffz1,acoffz2,deta,epsilu, 
                                     fsx, dsx, fsy, dsy,fsz,dsz, is, SV);
        s_u0=s_u1; s_v0=s_v1; s_w0=s_w1; s_px0=s_px1; s_py0=s_py1; s_pz0=s_pz1; s_qx0=s_qx1; s_qy0=s_qy1; s_qz0=s_qz1; 

        shot_record3d<<<(nx*ny+511)/512, 512>>>(nnx,nny, nnz, nx,ny, nz, npml, it, nt, s_P, shot_Dev);


           if((is==0)&&(it!=0&&it%300==0))
            {
	       hipMemcpy(e, s_P, nnz*nnx*nny*sizeof(float), hipMemcpyDeviceToHost);
              fseek(fpsnap,(int)(it/300-1)*(nx*ny*nz)*4L,0);
              window3d(v, e, nz, nx, ny, nnz, nnx, npml);
              fwrite(v,4L,nx*nz*ny,fpsnap);
            }
     }//it loop end
      mute_directwave3d<<<Xdimg,dimb>>>(nx,ny,nt,dt_,favg_,dx_,dy_,dz_,fsx,fsy,dsx,dsy,fsz,is,vp,epsilu,shot_Dev,100);
      hipMemcpy(shot_Hos, shot_Dev, nt*nx*ny*sizeof(float), hipMemcpyDeviceToHost);
      fseek(fpshot,is*nt*nx*ny*sizeof(float),0);
      fwrite(shot_Hos,sizeof(float),nt*nx*ny,fpshot);

    }//is loop end
    end = clock();
/*********IS Loop end*********/ 		     
   printf("---   The forward is over    \n"); 
   printf("---   Complete!!!!!!!!! \n");  
   printf("total %d shots: %f (s)\n", ns, ((float)(end-start))/CLOCKS_PER_SEC);



/***********close************/ 
          fclose(fpsnap);   fclose(fpshot);
/***********free*************/ 
       hipFree(coffx1);       hipFree(coffx2);
       hipFree(coffz1);       hipFree(coffz2);
       hipFree(acoffx1);      hipFree(acoffx2);
       hipFree(acoffz1);      hipFree(acoffz2);

       hipFree(s_u0);           hipFree(s_u1);
       hipFree(s_v0);           hipFree(s_v1);
       hipFree(s_w0);           hipFree(s_w1);

       hipFree(s_P);            hipFree(s_Q);

       hipFree(s_px0);          hipFree(s_px1);
       hipFree(s_py0);          hipFree(s_py1);
       hipFree(s_pz0);          hipFree(s_pz1);
       hipFree(s_qx0);          hipFree(s_qx1);
       hipFree(s_qy0);          hipFree(s_qy1);
       hipFree(s_qz0);          hipFree(s_qz1);

       hipFree(shot_Dev);
/***************host free*****************/
	free(v);	free(e);	free(d);
       free(shot_Hos);
}

